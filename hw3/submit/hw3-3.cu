#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <sys/mman.h>
#include <sys/stat.h> 
#include <sys/types.h>
#include <fcntl.h>
#include <unistd.h>
#include <omp.h>

#define B 64
#define B_half 32

const int INF = ((1 << 30) - 1);
int *Dist = NULL;
int n, m, N;

inline void input(char* infile);
inline void output(char* outFileName);
inline int ceil(int a, int b);
inline void block_FW();
__global__ void phase_one(int *dst, int Round, int N);
__global__ void phase_two(int *dst, int Round, int N);
__global__ void phase_three(int *dst, int Round, int N, int row_offset);

__device__ int Min(int a, int b) {
	return min(a, b);
}

int main(int argc, char* argv[]) {
	input(argv[1]);
	block_FW();
	output(argv[2]);
	return 0;
}

inline int ceil(int a, int b) { return (a + b - 1) / b; }

inline void block_FW() {
	int round = ceil(n, B);
	int* dst[2];

	hipHostRegister(Dist, N*N*sizeof(int), hipHostRegisterDefault);
	
	int blocks = (N + B - 1) / B;
	dim3 block_dim(32, 32);

	#pragma omp parallel num_threads(2)
	{
		// get thread number
		unsigned int cpu_thread_id = omp_get_thread_num();
		// thread neighbor number
		unsigned int cpu_thread_id_nei = !cpu_thread_id;

		// thread set its device and malloc same memory in the device
		hipSetDevice(cpu_thread_id);
		hipMalloc(&dst[cpu_thread_id], N*N*sizeof(int));
 
		unsigned int start_offset = (cpu_thread_id == 1) ? round / 2 : 0;
		unsigned int total_row = round / 2;
		if (round % 2 == 1 && cpu_thread_id == 1) total_row += 1;

		dim3 grid_dim(blocks, total_row);

		unsigned int dist_offset = start_offset * N * B;
		unsigned int total_byte_num = total_row * N * B * sizeof(int);
		unsigned int one_row_byte_num = B * N * sizeof(int);

		hipMemcpy(dst[cpu_thread_id] + dist_offset, Dist + dist_offset, total_byte_num, hipMemcpyHostToDevice);
		hipDeviceEnablePeerAccess(cpu_thread_id_nei, 0);
		#pragma omp barrier

		for (int r = 0; r < round; ++r) {
			unsigned int start_offset_num = r * B * N;
			if (r >= start_offset && r < (start_offset + total_row)) {
				hipMemcpy(dst[cpu_thread_id_nei] + start_offset_num, dst[cpu_thread_id] + start_offset_num, one_row_byte_num, hipMemcpyDefault);
			}
			#pragma omp barrier
			phase_one<<<1, block_dim>>>(dst[cpu_thread_id], r, N);
			phase_two<<<blocks, block_dim>>>(dst[cpu_thread_id], r, N);
			phase_three<<<grid_dim, block_dim>>>(dst[cpu_thread_id], r, N, start_offset);
		}
		hipMemcpy(Dist + dist_offset, dst[cpu_thread_id] + dist_offset, total_byte_num, hipMemcpyDeviceToHost);
	}
	hipFree(dst[0]);
	hipFree(dst[1]);
}

__global__ void phase_one(int *dst, int Round, int N) {
	int i = threadIdx.y;
	int j = threadIdx.x;

	int place = Round * B * (N + 1) + i * N + j;
	int place_right = Round * B * (N + 1) + i * N + j + B_half;
	int place_down = Round * B * (N + 1) + (i + B_half) * N + j;
	int place_down_right = Round * B * (N + 1) + (i + B_half) * N + j + B_half;

	__shared__ int s[B][B];

	// load gloabal data to local memory
	s[i][j] = dst[place];
	s[i][j+B_half] = dst[place_right];
	s[i+B_half][j] = dst[place_down];
	s[i+B_half][j+B_half] = dst[place_down_right];
	__syncthreads();

	for (int k = 0; k < B; ++k) {
		s[i][j] = Min(s[i][k] + s[k][j], s[i][j]);
		s[i][j+B_half] = Min(s[i][k] + s[k][j+B_half], s[i][j+B_half]);
		s[i+B_half][j] = Min(s[i+B_half][k] + s[k][j], s[i+B_half][j]);
		s[i+B_half][j+B_half] = Min(s[i+B_half][k] + s[k][j+B_half], s[i+B_half][j+B_half]);
		__syncthreads();
	}
	dst[place] = s[i][j];
	dst[place_right] = s[i][j+B_half];
	dst[place_down] = s[i+B_half][j];
	dst[place_down_right] = s[i+B_half][j+B_half];
}

__global__ void phase_two(int *dst, int Round, int N) {
	if (blockIdx.x == Round) return;

	int i = threadIdx.y;
	int j = threadIdx.x;

	int diagonal_place = Round * B * (N + 1) + i * N + j;
	int diagonal_place_right = Round * B * (N + 1) + i * N + j + B_half;
	int diagonal_place_down = Round * B * (N + 1) + (i + B_half) * N + j;
	int diagonal_place_down_right = Round * B * (N + 1) + (i + B_half) * N + j + B_half;

	int ver_place = blockIdx.x * B * N + Round * B + i * N + j;
	int ver_place_right = blockIdx.x * B * N + Round * B + i * N + j + B_half;
	int ver_place_down = blockIdx.x * B * N + Round * B + (i + B_half) * N + j;
	int ver_place_down_right = blockIdx.x * B * N + Round * B + (i + B_half) * N + j + B_half;

	int hor_place = Round * B * N + blockIdx.x * B + i * N + j;
	int hor_place_right = Round * B * N + blockIdx.x * B + i * N + j + B_half;
	int hor_place_down = Round * B * N + blockIdx.x * B + (i + B_half) * N + j;
	int hor_place_down_right = Round * B * N + blockIdx.x * B + (i + B_half) * N + j + B_half;

	__shared__ int s[B][B];
	__shared__ int ver[B][B];
	__shared__ int hor[B][B];

	s[i][j] = dst[diagonal_place];
	s[i][j+B_half] = dst[diagonal_place_right];
	s[i+B_half][j] = dst[diagonal_place_down];
	s[i+B_half][j+B_half] = dst[diagonal_place_down_right];

	ver[i][j] = dst[ver_place];
	ver[i][j+B_half] = dst[ver_place_right];
	ver[i+B_half][j] = dst[ver_place_down];
	ver[i+B_half][j+B_half] = dst[ver_place_down_right];

	hor[i][j] = dst[hor_place];
	hor[i][j+B_half] = dst[hor_place_right];
	hor[i+B_half][j] = dst[hor_place_down];
	hor[i+B_half][j+B_half] = dst[hor_place_down_right];

	__syncthreads();
	
	for (int k = 0; k < B; ++k) {
		ver[i][j] = Min(ver[i][j], ver[i][k] + s[k][j]);
		ver[i][j+B_half] = Min(ver[i][j+B_half], ver[i][k] + s[k][j+B_half]);
		ver[i+B_half][j] = Min(ver[i+B_half][j], ver[i+B_half][k] + s[k][j]);
		ver[i+B_half][j+B_half] = Min(ver[i+B_half][j+B_half], ver[i+B_half][k] + s[k][j+B_half]);

		hor[i][j] = Min(hor[i][j], s[i][k] + hor[k][j]);
		hor[i][j+B_half] = Min(hor[i][j+B_half], s[i][k] + hor[k][j+B_half]);
		hor[i+B_half][j] = Min(hor[i+B_half][j], s[i+B_half][k] + hor[k][j]);
		hor[i+B_half][j+B_half] = Min(hor[i+B_half][j+B_half], s[i+B_half][k] + hor[k][j+B_half]);

		__syncthreads();
	}

	dst[ver_place] = ver[i][j];
	dst[ver_place_right] = ver[i][j+B_half];
	dst[ver_place_down] = ver[i+B_half][j];
	dst[ver_place_down_right] = ver[i+B_half][j+B_half];

	dst[hor_place] = hor[i][j];
	dst[hor_place_right] = hor[i][j+B_half];
	dst[hor_place_down] = hor[i+B_half][j];
	dst[hor_place_down_right] = hor[i+B_half][j+B_half];
}

__global__ void phase_three(int *dst, int Round, int N, int row_offset) {
	if (blockIdx.x == Round || blockIdx.y + row_offset == Round) return;

	int block_y = blockIdx.y + row_offset;
	int i = threadIdx.y;
	int j = threadIdx.x;

	int self_place = block_y * B * N + blockIdx.x * B + i * N + j;
	int self_place_right = block_y * B * N + blockIdx.x * B + i * N + j + B_half;
	int self_place_down = block_y * B * N + blockIdx.x * B + (i + B_half) * N + j;
	int self_place_down_right = block_y * B * N + blockIdx.x * B + (i + B_half) * N + j + B_half;

	int a_place = block_y * B * N + Round * B + i * N + j;
	int a_place_right = block_y * B * N + Round * B + i * N + j + B_half;
	int a_place_down = block_y * B * N + Round * B + (i + B_half) * N + j;
	int a_place_down_right = block_y * B * N + Round * B + (i + B_half) * N + j + B_half;

	int b_place = Round * B * N + blockIdx.x * B + i * N + j;
	int b_place_right = Round * B * N + blockIdx.x * B + i * N + j + B_half;
	int b_place_down = Round * B * N + blockIdx.x * B + (i + B_half) * N + j;
	int b_place_down_right = Round * B * N + blockIdx.x * B + (i + B_half) * N + j + B_half;

	__shared__ int self[B][B];
	__shared__ int a[B][B];
	__shared__ int b[B][B];

	self[i][j] = dst[self_place];
	self[i][j+B_half] = dst[self_place_right];
	self[i+B_half][j] = dst[self_place_down];
	self[i+B_half][j+B_half] = dst[self_place_down_right];

	a[i][j] = dst[a_place];
	a[i][j+B_half] = dst[a_place_right];
	a[i+B_half][j] = dst[a_place_down];
	a[i+B_half][j+B_half] = dst[a_place_down_right];

	b[i][j] = dst[b_place];
	b[i][j+B_half] = dst[b_place_right];
	b[i+B_half][j] = dst[b_place_down];
	b[i+B_half][j+B_half] = dst[b_place_down_right];

	__syncthreads();

	#pragma unroll 32
	for (int k = 0; k < B; ++k) {
		self[i][j] = Min(a[i][k] + b[k][j], self[i][j]);
		self[i][j+B_half] = Min(a[i][k] + b[k][j+B_half], self[i][j+B_half]);
		self[i+B_half][j] = Min(a[i+B_half][k] + b[k][j], self[i+B_half][j]);
		self[i+B_half][j+B_half] = Min(a[i+B_half][k] + b[k][j+B_half], self[i+B_half][j+B_half]);
	}
	dst[self_place] = self[i][j];
	dst[self_place_right] = self[i][j+B_half];
	dst[self_place_down] = self[i+B_half][j];
	dst[self_place_down_right] = self[i+B_half][j+B_half];
}

inline void output(char* outFileName) {
	FILE* outfile = fopen(outFileName, "w");
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			if (Dist[i*N+j] >= INF) Dist[i*N+j] = INF;
		}
		fwrite(&Dist[i*N], sizeof(int), n, outfile);
	}
	fclose(outfile);
}

inline void input(char* infile) {
	int file = open(infile, O_RDONLY);
	int *ft = (int*)mmap(NULL, 2*sizeof(int), PROT_READ, MAP_PRIVATE, file, 0);
  n = ft[0];
	m = ft[1];
	int *pair = (int*)(mmap(NULL, (3 * m + 2) * sizeof(int), PROT_READ, MAP_PRIVATE, file, 0));

	if (n % B) N = n + (B - n % B);
	else N = n;

	Dist = (int*)malloc(sizeof(int)*N*N);
	

	for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
			Dist[i*N+j] = INF;
			if (i == j) Dist[i*N+j] = 0;
		}
  }

	#pragma unroll 4
	for (int i = 0; i < m; ++i) {
		Dist[pair[i*3+2]*N+pair[i*3+3]]= pair[i*3+4];
	}
	close(file);
	munmap(pair, (3 * m + 2) * sizeof(int));
}