#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define B 64
#define B_half 32

const int INF = ((1 << 30) - 1);
int *Dist = NULL;
int n, m, N;

inline void input(char* infile);
inline void output(char* outFileName);
inline int ceil(int a, int b);
inline void block_FW();
__global__ void phase_one(int *dst, int Round, int N);
__global__ void phase_two(int *dst, int Round, int N);
__global__ void phase_three(int *dst, int Round, int N);

__device__ int Min(int a, int b) {
	return min(a, b);
} 

int main(int argc, char* argv[]) {
	input(argv[1]);
	block_FW();
	output(argv[2]);
	return 0;
}

inline int ceil(int a, int b) { return (a + b - 1) / b; }

inline void block_FW() {
	int round = ceil(n, B);
	int *dst = NULL;
	hipMalloc(&dst, N*N*sizeof(int));
	hipMemcpy(dst, Dist, N*N*sizeof(int), hipMemcpyHostToDevice);
	
	int blocks = (N + B - 1) / B;
	dim3 block_dim(32, 32);
	dim3 grid_dim(blocks, blocks);
	for (int r = 0; r < round; ++r) {
		// phase 1
		phase_one<<<1, block_dim>>>(dst, r, N);
		// phase 2
		phase_two<<<blocks, block_dim>>>(dst, r, N);
		// phase 3
		phase_three<<<grid_dim, block_dim>>>(dst, r, N);
	}
	hipMemcpy(Dist, dst, N*N*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dst);
}

__global__ void phase_one(int *dst, int Round, int N) {
	int i = threadIdx.y;
	int j = threadIdx.x;

	int place = Round * B * (N + 1) + i * N + j;
	int place_right = Round * B * (N + 1) + i * N + j + B_half;
	int place_down = Round * B * (N + 1) + (i + B_half) * N + j;
	int place_down_right = Round * B * (N + 1) + (i + B_half) * N + j + B_half;

	__shared__ int s[B][B];

	// load gloabal data to local memory
	s[i][j] = dst[place];
	s[i][j+B_half] = dst[place_right];
	s[i+B_half][j] = dst[place_down];
	s[i+B_half][j+B_half] = dst[place_down_right];
	__syncthreads();

	for (int k = 0; k < B; ++k) {
		s[i][j] = Min(s[i][k] + s[k][j], s[i][j]);
		s[i][j+B_half] = Min(s[i][k] + s[k][j+B_half], s[i][j+B_half]);
		s[i+B_half][j] = Min(s[i+B_half][k] + s[k][j], s[i+B_half][j]);
		s[i+B_half][j+B_half] = Min(s[i+B_half][k] + s[k][j+B_half], s[i+B_half][j+B_half]);
		__syncthreads();
	}
	dst[place] = s[i][j];
	dst[place_right] = s[i][j+B_half];
	dst[place_down] = s[i+B_half][j];
	dst[place_down_right] = s[i+B_half][j+B_half];
}

__global__ void phase_two(int *dst, int Round, int N) {
	if (blockIdx.x == Round) return;

	int i = threadIdx.y;
	int j = threadIdx.x;

	int diagonal_place = Round * B * (N + 1) + i * N + j;
	int diagonal_place_right = Round * B * (N + 1) + i * N + j + B_half;
	int diagonal_place_down = Round * B * (N + 1) + (i + B_half) * N + j;
	int diagonal_place_down_right = Round * B * (N + 1) + (i + B_half) * N + j + B_half;

	int ver_place = blockIdx.x * B * N + Round * B + i * N + j;
	int ver_place_right = blockIdx.x * B * N + Round * B + i * N + j + B_half;
	int ver_place_down = blockIdx.x * B * N + Round * B + (i + B_half) * N + j;
	int ver_place_down_right = blockIdx.x * B * N + Round * B + (i + B_half) * N + j + B_half;

	int hor_place = Round * B * N + blockIdx.x * B + i * N + j;
	int hor_place_right = Round * B * N + blockIdx.x * B + i * N + j + B_half;
	int hor_place_down = Round * B * N + blockIdx.x * B + (i + B_half) * N + j;
	int hor_place_down_right = Round * B * N + blockIdx.x * B + (i + B_half) * N + j + B_half;

	__shared__ int s[B][B];
	__shared__ int ver[B][B];
	__shared__ int hor[B][B];

	s[i][j] = dst[diagonal_place];
	s[i][j+B_half] = dst[diagonal_place_right];
	s[i+B_half][j] = dst[diagonal_place_down];
	s[i+B_half][j+B_half] = dst[diagonal_place_down_right];

	ver[i][j] = dst[ver_place];
	ver[i][j+B_half] = dst[ver_place_right];
	ver[i+B_half][j] = dst[ver_place_down];
	ver[i+B_half][j+B_half] = dst[ver_place_down_right];

	hor[i][j] = dst[hor_place];
	hor[i][j+B_half] = dst[hor_place_right];
	hor[i+B_half][j] = dst[hor_place_down];
	hor[i+B_half][j+B_half] = dst[hor_place_down_right];

	__syncthreads();
	
	#pragma unroll
	for (int k = 0; k < B; ++k) {
		ver[i][j] = Min(ver[i][j], ver[i][k] + s[k][j]);
		ver[i][j+B_half] = Min(ver[i][j+B_half], ver[i][k] + s[k][j+B_half]);
		ver[i+B_half][j] = Min(ver[i+B_half][j], ver[i+B_half][k] + s[k][j]);
		ver[i+B_half][j+B_half] = Min(ver[i+B_half][j+B_half], ver[i+B_half][k] + s[k][j+B_half]);

		hor[i][j] = Min(hor[i][j], s[i][k] + hor[k][j]);
		hor[i][j+B_half] = Min(hor[i][j+B_half], s[i][k] + hor[k][j+B_half]);
		hor[i+B_half][j] = Min(hor[i+B_half][j], s[i+B_half][k] + hor[k][j]);
		hor[i+B_half][j+B_half] = Min(hor[i+B_half][j+B_half], s[i+B_half][k] + hor[k][j+B_half]);
	}

	dst[ver_place] = ver[i][j];
	dst[ver_place_right] = ver[i][j+B_half];
	dst[ver_place_down] = ver[i+B_half][j];
	dst[ver_place_down_right] = ver[i+B_half][j+B_half];

	dst[hor_place] = hor[i][j];
	dst[hor_place_right] = hor[i][j+B_half];
	dst[hor_place_down] = hor[i+B_half][j];
	dst[hor_place_down_right] = hor[i+B_half][j+B_half];
}

__global__ void phase_three(int *dst, int Round, int N) {
	if (blockIdx.x == Round || blockIdx.y == Round) return;

	int i = threadIdx.y;
	int j = threadIdx.x;

	int self_place = blockIdx.y * B * N + blockIdx.x * B + i * N + j;
	int self_place_right = blockIdx.y * B * N + blockIdx.x * B + i * N + j + B_half;
	int self_place_down = blockIdx.y * B * N + blockIdx.x * B + (i + B_half) * N + j;
	int self_place_down_right = blockIdx.y * B * N + blockIdx.x * B + (i + B_half) * N + j + B_half;

	int a_place = blockIdx.y * B * N + Round * B + i * N + j;
	int a_place_right = blockIdx.y * B * N + Round * B + i * N + j + B_half;
	int a_place_down = blockIdx.y * B * N + Round * B + (i + B_half) * N + j;
	int a_place_down_right = blockIdx.y * B * N + Round * B + (i + B_half) * N + j + B_half;

	int b_place = Round * B * N + blockIdx.x * B + i * N + j;
	int b_place_right = Round * B * N + blockIdx.x * B + i * N + j + B_half;
	int b_place_down = Round * B * N + blockIdx.x * B + (i + B_half) * N + j;
	int b_place_down_right = Round * B * N + blockIdx.x * B + (i + B_half) * N + j + B_half;

	__shared__ int self[B][B];
	__shared__ int a[B][B];
	__shared__ int b[B][B];

	self[i][j] = dst[self_place];
	self[i][j+B_half] = dst[self_place_right];
	self[i+B_half][j] = dst[self_place_down];
	self[i+B_half][j+B_half] = dst[self_place_down_right];

	a[i][j] = dst[a_place];
	a[i][j+B_half] = dst[a_place_right];
	a[i+B_half][j] = dst[a_place_down];
	a[i+B_half][j+B_half] = dst[a_place_down_right];

	b[i][j] = dst[b_place];
	b[i][j+B_half] = dst[b_place_right];
	b[i+B_half][j] = dst[b_place_down];
	b[i+B_half][j+B_half] = dst[b_place_down_right];

	__syncthreads();

	#pragma unroll(32)
	for (int k = 0; k < B; ++k) {
		self[i][j] = Min(a[i][k] + b[k][j], self[i][j]);
		self[i][j+B_half] = Min(a[i][k] + b[k][j+B_half], self[i][j+B_half]);
		self[i+B_half][j] = Min(a[i+B_half][k] + b[k][j], self[i+B_half][j]);
		self[i+B_half][j+B_half] = Min(a[i+B_half][k] + b[k][j+B_half], self[i+B_half][j+B_half]);
	}
	dst[self_place] = self[i][j];
	dst[self_place_right] = self[i][j+B_half];
	dst[self_place_down] = self[i+B_half][j];
	dst[self_place_down_right] = self[i+B_half][j+B_half];
}

inline void output(char* outFileName) {
	FILE* outfile = fopen(outFileName, "w");
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			if (Dist[i*N+j] >= INF) Dist[i*N+j] = INF;
		}
		fwrite(&Dist[i*N], sizeof(int), n, outfile);
	}
	fclose(outfile);
}

inline void input(char* infile) {
	FILE* file = fopen(infile, "rb");
	fread(&n, sizeof(int), 1, file);
	fread(&m, sizeof(int), 1, file);

	if (n % B) N = n + (B - n % B);
	else N = n;
	hipHostAlloc(&Dist, N*N*sizeof(int), 1);

	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
			if (i == j) {
				Dist[i*N+j] = 0;
			} else {
				Dist[i*N+j] = INF;
			}
		}
	}

	int pair[3];
	for (int i = 0; i < m; ++i) {
		fread(pair, sizeof(int), 3, file);
		Dist[pair[0]*N+pair[1]]= pair[2];
	}

	fclose(file);
}