#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <sys/mman.h>
#include <sys/stat.h> 
#include <sys/types.h>
#include <fcntl.h>
#include <unistd.h>
#define B 64
#define B_half 32

const int INF = ((1 << 30) - 1);
int *Dist = NULL;
int n, m, N;

inline void input(char* infile);
inline void output(char* outFileName);
inline int ceil(int a, int b);
inline void block_FW();
inline void floyed_warshall();
__global__ void phase_one(int *dst, int Round, int N);
__global__ void phase_two(int *dst, int Round, int N);
__global__ void phase_three(int *dst, int Round, int N);

__device__ int Min(int a, int b) {
	return min(a, b);
} 

int main(int argc, char* argv[]) {
	input(argv[1]);
	if (n <= 500) floyed_warshall();
	else block_FW();
	output(argv[2]);
	return 0;
}

inline void floyed_warshall() {
	for (int k = 0; k < n; k++) {
		for (int i = 0; i < n; i++) {
			for (int j = 0; j < n; j++) {
				if (Dist[i*N+j] > Dist[i*N+k] + Dist[k*N+j]) {
					Dist[i*N+j] = Dist[i*N+k] + Dist[k*N+j];
				}
			}
		}
	}
}

inline int ceil(int a, int b) { return (a + b - 1) / b; }

inline void block_FW() {
	int round = ceil(n, B);
	int *dst = NULL;
	unsigned int size = N*N*sizeof(int);
	hipHostRegister(Dist, size, hipHostRegisterDefault);
	hipMalloc(&dst, size);
	hipMemcpy(dst, Dist, size, hipMemcpyHostToDevice);
	
	int blocks = (N + B - 1) / B;
	dim3 block_dim(32, 32);
	dim3 grid_dim(blocks, blocks);
	for (int r = 0; r < round; ++r) {
		// phase 1
		phase_one<<<1, block_dim>>>(dst, r, N);
		// phase 2
		phase_two<<<blocks, block_dim>>>(dst, r, N);
		// phase 3
		phase_three<<<grid_dim, block_dim>>>(dst, r, N);
	}
	hipMemcpy(Dist, dst, N*N*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dst);
}

__global__ void phase_one(int *dst, int Round, int N) {
	int i = threadIdx.y;
	int j = threadIdx.x;

	int place = Round * B * (N + 1) + i * N + j;
	int place_right = Round * B * (N + 1) + i * N + j + B_half;
	int place_down = Round * B * (N + 1) + (i + B_half) * N + j;
	int place_down_right = Round * B * (N + 1) + (i + B_half) * N + j + B_half;

	int i_B = i + B_half;
	int j_B = j + B_half;

	__shared__ int s[B][B];

	// load gloabal data to local memory
	s[i][j] = dst[place];
	s[i][j_B] = dst[place_right];
	s[i_B][j] = dst[place_down];
	s[i_B][j_B] = dst[place_down_right];
	__syncthreads();

	for (int k = 0; k < B; ++k) {
		s[i][j] = Min(s[i][k] + s[k][j], s[i][j]);
		s[i][j_B] = Min(s[i][k] + s[k][j_B], s[i][j_B]);
		s[i_B][j] = Min(s[i_B][k] + s[k][j], s[i_B][j]);
		s[i_B][j_B] = Min(s[i_B][k] + s[k][j_B], s[i_B][j_B]);
		__syncthreads();
	}
	dst[place] = s[i][j];
	dst[place_right] = s[i][j_B];
	dst[place_down] = s[i_B][j];
	dst[place_down_right] = s[i_B][j_B];
}

__global__ void phase_two(int *dst, int Round, int N) {
	if (blockIdx.x == Round) return;

	int i = threadIdx.y;
	int j = threadIdx.x;

	int diagonal_place = Round * B * (N + 1) + i * N + j;
	int diagonal_place_right = Round * B * (N + 1) + i * N + j + B_half;
	int diagonal_place_down = Round * B * (N + 1) + (i + B_half) * N + j;
	int diagonal_place_down_right = Round * B * (N + 1) + (i + B_half) * N + j + B_half;

	int ver_place = blockIdx.x * B * N + Round * B + i * N + j;
	int ver_place_right = blockIdx.x * B * N + Round * B + i * N + j + B_half;
	int ver_place_down = blockIdx.x * B * N + Round * B + (i + B_half) * N + j;
	int ver_place_down_right = blockIdx.x * B * N + Round * B + (i + B_half) * N + j + B_half;

	int hor_place = Round * B * N + blockIdx.x * B + i * N + j;
	int hor_place_right = Round * B * N + blockIdx.x * B + i * N + j + B_half;
	int hor_place_down = Round * B * N + blockIdx.x * B + (i + B_half) * N + j;
	int hor_place_down_right = Round * B * N + blockIdx.x * B + (i + B_half) * N + j + B_half;

	int i_B = i + B_half;
	int j_B = j + B_half;

	__shared__ int s[B][B];
	__shared__ int ver[B][B];
	__shared__ int hor[B][B];

	s[i][j] = dst[diagonal_place];
	s[i][j_B] = dst[diagonal_place_right];
	s[i_B][j] = dst[diagonal_place_down];
	s[i_B][j_B] = dst[diagonal_place_down_right];

	ver[i][j] = dst[ver_place];
	ver[i][j_B] = dst[ver_place_right];
	ver[i_B][j] = dst[ver_place_down];
	ver[i_B][j_B] = dst[ver_place_down_right];

	hor[i][j] = dst[hor_place];
	hor[i][j_B] = dst[hor_place_right];
	hor[i_B][j] = dst[hor_place_down];
	hor[i_B][j_B] = dst[hor_place_down_right];

	__syncthreads();
	
	for (int k = 0; k < B; ++k) {
		ver[i][j] = Min(ver[i][j], ver[i][k] + s[k][j]);
		ver[i][j_B] = Min(ver[i][j_B], ver[i][k] + s[k][j_B]);
		ver[i_B][j] = Min(ver[i_B][j], ver[i_B][k] + s[k][j]);
		ver[i_B][j_B] = Min(ver[i_B][j_B], ver[i_B][k] + s[k][j_B]);

		hor[i][j] = Min(hor[i][j], s[i][k] + hor[k][j]);
		hor[i][j_B] = Min(hor[i][j_B], s[i][k] + hor[k][j_B]);
		hor[i_B][j] = Min(hor[i_B][j], s[i_B][k] + hor[k][j]);
		hor[i_B][j_B] = Min(hor[i_B][j_B], s[i_B][k] + hor[k][j_B]);
		
		__syncthreads();
	}

	dst[ver_place] = ver[i][j];
	dst[ver_place_right] = ver[i][j_B];
	dst[ver_place_down] = ver[i_B][j];
	dst[ver_place_down_right] = ver[i_B][j_B];

	dst[hor_place] = hor[i][j];
	dst[hor_place_right] = hor[i][j_B];
	dst[hor_place_down] = hor[i_B][j];
	dst[hor_place_down_right] = hor[i_B][j_B];
}

__global__ void phase_three(int *dst, int Round, int N) {
	if (blockIdx.x == Round || blockIdx.y == Round) return;

	int i = threadIdx.y;
	int j = threadIdx.x;

	int self_place = blockIdx.y * B * N + blockIdx.x * B + i * N + j;
	int self_place_right = blockIdx.y * B * N + blockIdx.x * B + i * N + j + B_half;
	int self_place_down = blockIdx.y * B * N + blockIdx.x * B + (i + B_half) * N + j;
	int self_place_down_right = blockIdx.y * B * N + blockIdx.x * B + (i + B_half) * N + j + B_half;

	int a_place = blockIdx.y * B * N + Round * B + i * N + j;
	int a_place_right = blockIdx.y * B * N + Round * B + i * N + j + B_half;
	int a_place_down = blockIdx.y * B * N + Round * B + (i + B_half) * N + j;
	int a_place_down_right = blockIdx.y * B * N + Round * B + (i + B_half) * N + j + B_half;

	int b_place = Round * B * N + blockIdx.x * B + i * N + j;
	int b_place_right = Round * B * N + blockIdx.x * B + i * N + j + B_half;
	int b_place_down = Round * B * N + blockIdx.x * B + (i + B_half) * N + j;
	int b_place_down_right = Round * B * N + blockIdx.x * B + (i + B_half) * N + j + B_half;

	int i_B = i + B_half;
	int j_B = j + B_half;

	__shared__ int self[B][B];
	__shared__ int a[B][B];
	__shared__ int b[B][B];

	self[i][j] = dst[self_place];
	self[i][j_B] = dst[self_place_right];
	self[i_B][j] = dst[self_place_down];
	self[i_B][j_B] = dst[self_place_down_right];

	a[i][j] = dst[a_place];
	a[i][j_B] = dst[a_place_right];
	a[i_B][j] = dst[a_place_down];
	a[i_B][j_B] = dst[a_place_down_right];

	b[i][j] = dst[b_place];
	b[i][j_B] = dst[b_place_right];
	b[i_B][j] = dst[b_place_down];
	b[i_B][j_B] = dst[b_place_down_right];

	__syncthreads();

	#pragma unroll 32
	for (int k = 0; k < B; ++k) {
		self[i][j] = Min(a[i][k] + b[k][j], self[i][j]);
		self[i][j_B] = Min(a[i][k] + b[k][j_B], self[i][j_B]);
		self[i_B][j] = Min(a[i_B][k] + b[k][j], self[i_B][j]);
		self[i_B][j_B] = Min(a[i_B][k] + b[k][j_B], self[i_B][j_B]);
	}
	dst[self_place] = self[i][j];
	dst[self_place_right] = self[i][j_B];
	dst[self_place_down] = self[i_B][j];
	dst[self_place_down_right] = self[i_B][j_B];
}

inline void output(char* outFileName) {
	FILE* outfile = fopen(outFileName, "w");

	#pragma unroll 32
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			if (Dist[i*N+j] >= INF) Dist[i*N+j] = INF;
		}
		fwrite(&Dist[i*N], sizeof(int), n, outfile);
	}
	fclose(outfile);
}

inline void input(char* infile) {
	int file = open(infile, O_RDONLY);
	int *ft = (int*)mmap(NULL, 2*sizeof(int), PROT_READ, MAP_PRIVATE, file, 0);
  n = ft[0];
	m = ft[1];
	int *pair = (int*)(mmap(NULL, (3 * m + 2) * sizeof(int), PROT_READ, MAP_PRIVATE, file, 0));

	if (n % B) N = n + (B - n % B);
	else N = n;

	Dist = (int*)malloc(N*N*sizeof(int));
	

	for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
			Dist[i*N+j] = INF;
			if (i == j) Dist[i*N+j] = 0;
		}
  }

	#pragma unroll
	for (int i = 0; i < m; ++i) {
		Dist[pair[i*3+2]*N+pair[i*3+3]]= pair[i*3+4];
	}
	close(file);
	munmap(pair, (3 * m + 2) * sizeof(int));
}